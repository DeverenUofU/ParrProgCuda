
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#define threshold 0.0001
#define FIXME 1


void checkCUDAError(const char *msg);

const int DSIZE = 1024;
hipEvent_t start, stop;
float tstart, elapsedTime;

// matrix multiply kernel: C = A * B
__global__ void mmtt_i4(const float *A, const float *B, float *C, int ds) {

    int tx = threadIdx.x;
    int i = (4 * blockIdx.x) * blockDim.x + tx;  //now to change to multiply for increased stride
    int j = blockIdx.y;                 

    if (i < ds && j < ds) {
        float sum0 = 0.0;
        float sum1 = 0.0;
        float sum2 = 0.0;
        float sum3 = 0.0;
        for (int k = 0; k < ds; ++k) {
            sum0 += A[i + ds*k] * B[k + ds*j];
            sum1 += A[(i+blockIdx.x) + ds*k] * B[k + ds*j];
            sum2 += A[(i+blockIdx.x+blockIdx.x) + ds*k] * B[k + ds*j];
            sum3 += A[(i+blockIdx.x+blockIdx.x+blockIdx.x) + ds*k] * B[k + ds*j];
        }
        C[i * ds + j] = sum0;
        C[(i+blockIdx.x) * ds + j] = sum1;
        C[(i+blockIdx.x+blockIdx.x) * ds + j] = sum2;
        C[(i+blockIdx.x+blockIdx.x+blockIdx.x) * ds + j] = sum3;
    }
}

int main(){

  float *h_A, *h_B, *h_C, *h_Cref, *d_A, *d_B, *d_C;
  int i,j,k;

  h_A = new float[DSIZE*DSIZE];
  h_B = new float[DSIZE*DSIZE];
  h_C = new float[DSIZE*DSIZE];
  h_Cref = new float[DSIZE*DSIZE];
  for (i = 0; i < DSIZE*DSIZE; i++){
    h_A[i] = rand();
    h_B[i] = rand();
    h_C[i] = 0;
    h_Cref[i] = 0;}

  for (i=0;i<DSIZE;i++)
   for (k=0;k<DSIZE;k++)
    for (j=0;j<DSIZE;j++)
//   h_Cref[i][j] += h_A[k][i]*h_B[j][k];
     h_Cref[i*DSIZE+j] += h_A[k*DSIZE+i]*h_B[j*DSIZE+k];
  
 // Allocate device memory and copy input data over to GPU
  hipMalloc(&d_A, DSIZE*DSIZE*sizeof(float));
  hipMalloc(&d_B, DSIZE*DSIZE*sizeof(float));
  hipMalloc(&d_C, DSIZE*DSIZE*sizeof(float));
  checkCUDAError("hipMalloc failure");
  hipMemcpy(d_A, h_A, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy H2D transfer failure");


  dim3 block(256,1);  
//   dim3 grid((DSIZE + block.x - 1) / block.x, (DSIZE + 4 - 1) / 4); // Adjusted for i-loop unrolling
  dim3 grid(((DSIZE + block.x - 1) / 4) / block.x, DSIZE);    // Adjusted for x dim,  i-loop unrolling
  printf("Matrix size: %d\n", DSIZE);


  for(int trial=0;trial<3;trial++)
  {
   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEventRecord(start);
   // Launch kernel
   mmtt_i4<<<grid, block>>>(d_A, d_B, d_C, DSIZE);
   checkCUDAError("GPU kernel launch failure");
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&elapsedTime, start,stop);
   hipDeviceSynchronize();
   // Copy results back to host
   hipMemcpy(h_C, d_C, DSIZE*DSIZE*sizeof(float), hipMemcpyDeviceToHost);
   checkCUDAError("hipMemcpy D2H");
   for (int i = 0; i < DSIZE*DSIZE; i++) if (fabs((h_C[i]-h_Cref[i])/h_Cref[i])>threshold) {printf("Error: mismatch at linearized index %d, was: %f, should be: %f\n", i, h_C[i], h_Cref[i]); return -1;}
   printf("Trial %d: GFLOPS: %.2f\n",trial,2.0e-6*DSIZE*DSIZE*DSIZE/elapsedTime);
  }
  return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

